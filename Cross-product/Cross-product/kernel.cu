#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define CHECK(_t, _e) if (_e != hipSuccess) { fprintf(stderr, "%s failed: %s", _t, hipGetErrorString(_e)); goto Error;}
#define HERR(_t, _e) if (_e != hipSuccess) { fprintf(stderr, "%s failed: %s", _t, hipGetErrorString(_e));}

const int len = 2 * 1024 + 5;

__global__ void kernel(float *c, float *a, float *b)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < len) {
		c[i] = a[i] * b[i];
	}
}

int main()
{
	int i, ndev, bc, tc;
	hipDeviceProp_t p;

	float* a = (float*)malloc(len * sizeof(float));
	float* b = (float*)malloc(len * sizeof(float));
	float* c = (float*)malloc(len * sizeof(float));

	float* da = NULL;
	float* db = NULL;
	float* dc = NULL;

	for (i = 0; i < len; i++) {
		a[i] = 0.5f;
		b[i] = 2.0f;
	}

	//print device properties
	CHECK("hipGetDeviceCount", hipGetDeviceCount(&ndev));
	for (i = 0; i < ndev; i++) {
		CHECK("hipGetDeviceProperties", hipGetDeviceProperties(&p, i));
		printf("Name: %s\n", p.name);
		printf("Compute capability: %d.%d\n", p.major, p.minor);
		printf("Max threads/block: %d\n", p.maxThreadsPerBlock);
		printf("Max block size: %d x %d x %d\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
		printf("Max grid size: %d x %d x %d\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
	}

	CHECK("hipSetDevice", hipSetDevice(0));

	CHECK("hipMalloc da", hipMalloc(&da, len * sizeof(float)));
	CHECK("hipMalloc db", hipMalloc(&db, len * sizeof(float)));
	CHECK("hipMalloc dc", hipMalloc(&dc, len * sizeof(float)));

	//transfer the data

	CHECK("hipMemcpy da", hipMemcpy(da, a, len * sizeof(float), hipMemcpyHostToDevice));
	CHECK("hipMemcpy db", hipMemcpy(db, b, len * sizeof(float), hipMemcpyHostToDevice));

	tc = 1024;
	bc = len / tc;
	if (len % tc != 0) {
		bc++;
	}

	kernel <<<bc, tc >>> (dc, da, db);
	CHECK("kernel", hipGetLastError());

	CHECK("hipMemcpy dc", hipMemcpy(c, dc, len * sizeof(float), hipMemcpyDeviceToHost));

	for (i = 0; i < len; i++) {
		if (i % 20 == 0) {
			printf("\n");
		}
		printf("% 2.0f", c[i]);
	}
	printf("\n");


Error:

	HERR("hipFree da", hipFree(da));
	HERR("hipFree db", hipFree(db));
	HERR("hipFree dc", hipFree(dc));
	HERR("hipDeviceReset", hipDeviceReset());

	return 0;
}
